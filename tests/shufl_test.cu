
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void shfl_down_example(int *data_out) {
    int tid = threadIdx.x;
    int lane = tid % 32;  // 当前线程在 warp 内的索引
    int warpId = tid / 32; // warp 的索引
    int sub_lane = lane % 8;

    // 每个线程的初始值为自己的线程索引
    int value = tid;

    // 使用 __shfl_down_sync 传递值，每个线程向下移动1位的值
    unsigned mask = 0xFFFFFFFF; // 全部线程都激活
    int shuffled = 0;
    shuffled = __shfl_down_sync(mask, value, 4, 8);

    shuffled = (sub_lane / 4 != 0)?  0 : shuffled;

    // 将 shuffle 后的值存入输出数组
    data_out[tid] = shuffled;
}

int main() {
    const int N = 32;  // 一个 warp 的线程数
    int h_data_out[N];

    // 分配 GPU 内存
    int *d_data_out;
    hipMalloc(&d_data_out, N * sizeof(int));

    // 执行核函数
    shfl_down_example<<<1, N>>>(d_data_out);

    // 将结果从设备复制到主机
    hipMemcpy(h_data_out, d_data_out, N * sizeof(int), hipMemcpyDeviceToHost);

    // 打印结果
    printf("Shuffled values (delta=1):\n");
    for (int i = 0; i < N; i++) {
        printf("Thread %2d: %2d\n", i, h_data_out[i]);
    }

    // 释放 GPU 内存
    hipFree(d_data_out);

    return 0;
}