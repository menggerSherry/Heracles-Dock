
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>
#include "error.cuh"
#include <iostream>




void eigh_gpu(double * eigenvalues, double * eigenvectors, double * matrix, int * eigvals, int n){


    hipsolverHandle_t cusolver_handle = NULL;
    hipStream_t stream = NULL;
    
    int lda = n;

    double * d_eigenvalues = nullptr;
    double * d_matrix = nullptr;
    double * d_work = nullptr;
    int *d_info = nullptr;

    int info = 0;
    int lwork = 0;
    // double * eigenvalues_buffer = new double [n];
    double * eigenvectors_buffer = new double [n*n];
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolver_handle));
    CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolver_handle, stream));



    CHECK(hipMalloc((void**)&d_matrix, n * n * sizeof(double)));
    CHECK(hipMalloc((void**)&d_eigenvalues,n*sizeof(double)));
    CHECK(hipMalloc((void**)&d_info,sizeof(int)));

    CHECK(hipMemcpyAsync(d_matrix, matrix, n * n * sizeof(double), hipMemcpyHostToDevice,stream));

    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    CUSOLVER_CHECK(hipsolverDnDsyevd_bufferSize(cusolver_handle,jobz,uplo,n,d_matrix,lda,d_eigenvalues,&lwork));

    CHECK(hipMalloc((void**)&d_work,sizeof(double)*lwork));

    // compute spectrum

    CUSOLVER_CHECK(hipsolverDnDsyevd(cusolver_handle, jobz, uplo, n, d_matrix, lda, d_eigenvalues, d_work, lwork, d_info));
    // check_cusolver(status);
    // std::printf("after syevd: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    int start = eigvals[0];
    int end = eigvals[1];
    int length = end - start + 1;
    CHECK(hipMemcpyAsync(eigenvectors_buffer, d_matrix, n * n * sizeof(double), hipMemcpyDeviceToHost,stream));
    CHECK(hipMemcpyAsync(eigenvalues, d_eigenvalues+start, length*sizeof(double),hipMemcpyDeviceToHost,stream));
    CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));
    CHECK(hipStreamSynchronize(stream));
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolver_handle));
    CHECK(hipFree(d_matrix));
    CHECK(hipFree(d_eigenvalues));
    CHECK(hipFree(d_work));

    // 

    for(int i = 0; i< length; i++){
        for(int j = 0; j< n; j++){
            eigenvectors[j*length+i] = eigenvectors_buffer[i*n+j];
        }
    }
    
    // for(int i = 0;i<n;i++){
    //     int index = 0;
    //     for(int j = start; j<=end; j++){
    //         eigenvectors[i*length+index] = eigenvectors_buffer[i*n+j];
    //         index++;
    //     }
    // }
    delete [] eigenvectors_buffer;

}

void build_hessian1D(double *hessian, double *kirchhoff, double cutoff, double gamma, int n_atoms, double *coords){
    double cutoff2 = cutoff * cutoff;
    int dof = n_atoms * 3;

    for (int i = 0; i < n_atoms; i++) {
        int res_i3 = i * 3;
        int i_p1 = i + 1;
        double *coord_i = &coords[i * 3];

        for (int j = i_p1; j < n_atoms; j++) {
            int res_j3 = j * 3;

            double i2j_all[3] = {
                coords[j * 3] - coord_i[0],
                coords[j * 3 + 1] - coord_i[1],
                coords[j * 3 + 2] - coord_i[2]
            };

            double dist2 = 0.0;
            for (int k = 0; k < 3; k++) {
                dist2 += i2j_all[k] * i2j_all[k];
            }

            if (dist2 > cutoff2) {
                continue;
            }

            double g = gamma;

            double super_element[3][3];
            for (int k = 0; k < 3; k++) {
                for (int l = 0; l < 3; l++) {
                    super_element[k][l] = i2j_all[k] * i2j_all[l] * (-g / dist2);
                }
            }

            for (int k = 0; k < 3; k++) {
                for (int l = 0; l < 3; l++) {
                    // 压缩 Hessian 矩阵为一维数组
                    hessian[(res_i3 + k) * dof + (res_j3 + l)] = super_element[k][l];
                    hessian[(res_j3 + l) * dof + (res_i3 + k)] = super_element[k][l];
                    hessian[(res_i3 + k) * dof + (res_i3 + l)] -= super_element[k][l];
                    hessian[(res_j3 + l) * dof + (res_j3 + k)] -= super_element[k][l];
                }
            }

            kirchhoff[i * n_atoms + j] = -g;
            kirchhoff[j * n_atoms + i] = -g;
            kirchhoff[i * n_atoms + i] += g;
            kirchhoff[j * n_atoms + j] += g;
        }
    }
}


int main(){
    double A[9] = {4,2,1,2,5,3,1,3,6};
    double * val = (double *)malloc(3*sizeof(double));
    double * ei_vec = (double *)malloc(3*2*sizeof(double));
    int eival[2] = {0,1};
    eigh_gpu(val,ei_vec,A,eival,3);
    for(int i = 0; i<3;i++){
        for(int j = 0; j<2;j++){
            std::cout<<ei_vec[i*2+j]<<" ";
        }
        std::cout<<std::endl;
    }

    double coord[9] = {
        3,3,4,
        4.5,4.5,3.3,
        6.5,4.4,5.4,
    };
    double * hession = new double[9*9];
    double * kirchhoff = new double[3*3];

    build_hessian1D(hession,kirchhoff,15,1,3,coord);
    for(int i = 0; i<9;i++){
        for(int j = 0; j<9;j++){
            std::cout<<hession[i*9+j]<<" ";
        }
        std::cout<<std::endl;
    }

    return 0;
}



