#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA 内核函数，用于矩阵转置
__global__ void matrixTranspose(float* input, float* output) {
    int tid = threadIdx.x; // 获取当前线程 ID
    int row = tid / 4;     // 计算行号 (0~7)
    int col = tid % 4;     // 计算列号 (0~3)

    // 每个线程读取一个输入矩阵中的元素
    float value = input[row * 4 + col];

    // 通过 shfl 操作在线程间传递数据，实现矩阵转置

    value = __shfl_down_sync(0xFFFFFFFF, value, 7,16);
    value = __shfl_up_sync(0xFFFFFFFF, value, 4,16);
    value = __shfl_up_sync(0xFFFFFFFF, value, 8,16);
    value = __shfl_up_sync(0xFFFFFFFF, value, 2,4);
    
    value = __shfl_down_sync(0xFFFFFFFF, value, 14,32);
    value = __shfl_up_sync(0xFFFFFFFF, value, 4,8);
    value = __shfl_up_sync(0xFFFFFFFF, value, 8,16);
    value = __shfl_up_sync(0xFFFFFFFF, value, 16,32);


    // 写入转置后的矩阵到输出
    output[row * 4 + col] = value;
}

// 检查 CUDA 错误的辅助函数
void checkCudaError(hipError_t error, const char* msg) {
    if (error != hipSuccess) {
        printf("%s: %s\n", msg, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

// 打印矩阵
void printMatrix(const float* matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%5.2f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}

int main() {
    const int rows = 8;
    const int cols = 4;
    const int matrixSize = rows * cols * sizeof(float);
    
    // 定义并初始化主机端的输入矩阵
    float h_input[rows * cols];
    float h_output[rows * cols];

    // 初始化矩阵数据
    printf("Input Matrix:\n");
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            h_input[i * cols + j] = i * cols + j + 1; // 举例: 1, 2, 3, ..., 32
        }
    }
    printMatrix(h_input, rows, cols);

    // 定义设备端的输入输出矩阵
    float* d_input;
    float* d_output;

    // 分配设备端内存
    checkCudaError(hipMalloc(&d_input, matrixSize), "Failed to allocate device memory for input");
    checkCudaError(hipMalloc(&d_output, matrixSize), "Failed to allocate device memory for output");

    // 将输入矩阵从主机复制到设备
    checkCudaError(hipMemcpy(d_input, h_input, matrixSize, hipMemcpyHostToDevice), "Failed to copy input matrix to device");

    // 启动 kernel，假设一个 warp 32 个线程，每个线程一个 value
    matrixTranspose<<<1, 32>>>(d_input, d_output);

    // 等待 GPU 完成
    checkCudaError(hipDeviceSynchronize(), "Kernel execution failed");

    // 将结果从设备端复制回主机
    checkCudaError(hipMemcpy(h_output, d_output, matrixSize, hipMemcpyDeviceToHost), "Failed to copy output matrix to host");

    // 打印转置后的矩阵
    printf("Transposed Matrix:\n");
    printMatrix(h_output, rows, cols); // 打印转置矩阵

    // 释放设备内存
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
