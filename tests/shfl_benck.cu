#include <stdio.h>
#include <hip/hip_runtime.h>

#define ITERATIONS 1

// Kernel function to measure __shfl_sync latency in cycles
__global__ void benchmark_shfl_sync_cycles(unsigned long long *output_cycles) {
    volatile int value = threadIdx.x;
    int laneId = threadIdx.x % 32;

    // Record start clock
    unsigned long long start = clock64();

    
    value = __shfl_sync(0xffffffff, value, laneId, 4);
    

    // Record end clock
    unsigned long long end = clock64();

    // Return the number of cycles
    output_cycles[threadIdx.x] = (end - start);
}

// Kernel function to measure __shfl_down_sync latency in cycles
__global__ void benchmark_shfl_down_sync_cycles(unsigned long long *output_cycles) {
    volatile int value = threadIdx.x;

    // Record start clock
    unsigned long long start = clock64();

    
    value = __shfl_down_sync(0xffffffff, value, 1, 4);
    

    // Record end clock
    unsigned long long end = clock64();

    // Return the number of cycles
    output_cycles[threadIdx.x] = (end - start);
}

// Utility function to print the results
void print_cycles(const char* kernel_name, unsigned long long* h_output) {
    printf("%s cycles per warp:\n", kernel_name);
    for (int i = 0; i < 32; ++i) {
        printf("Thread %d: %llu cycles\n", i, h_output[i]);
    }
    printf("\n");
}

int main() {
    unsigned long long h_output[32];
    unsigned long long *d_output;

    hipMalloc(&d_output, 32 * sizeof(unsigned long long));

    // Measure cycles for __shfl_sync
    benchmark_shfl_sync_cycles<<<1, 32>>>(d_output);
    hipMemcpy(h_output, d_output, 32 * sizeof(unsigned long long), hipMemcpyDeviceToHost);
    print_cycles("__shfl_sync", h_output);

    // Measure cycles for __shfl_down_sync
    benchmark_shfl_down_sync_cycles<<<1, 32>>>(d_output);
    hipMemcpy(h_output, d_output, 32 * sizeof(unsigned long long), hipMemcpyDeviceToHost);
    print_cycles("__shfl_down_sync", h_output);

    // Cleanup
    hipFree(d_output);

    return 0;
}
