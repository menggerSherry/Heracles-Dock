#include "hip/hip_runtime.h"
#include <mma.h>
using namespace nvcuda;

const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

__global__ void wmma_example(half *a, half *b, float *c, 
                             int M, int N, int K, 
                             float alpha, float beta)
{
    int lda = M;
    int ldb = K;
    int ldc = M;
    int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    int warpN = (blockIdx.y * blockDim.y + threadIdx.y);
    // MMA是warp维度的操作，寄存器分布在warp中的线程中，每一个线程包邮一个片段
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_k, half, wmma::col_major> b_frag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K,  float> acc_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
    // a M K ; b K N
    for(int i = 0; i<k; i+=WMMA_K){
        int aRow = warpM * WMMA_M;
        int aCol = i;
        int bRow = i;
        int bCol = warpN * warpN;
        if(aRow < M && aCol < K && bRow<K && bCol < N){
            wmma::load_matrix_sync(a_frag , a+ aRow + aCol * lda, lda);
            wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);
            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
        }
    }
}


