#include "cal_anm.h"
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>
#include "error.cuh"





void eigh_gpu(double * eigenvalues, double * eigenvectors, double * matrix, int * eigvals, int n){


    hipsolverHandle_t cusolver_handle = NULL;
    // hipStream_t stream = NULL;
    
    int lda = n;

    double * d_eigenvalues = nullptr;
    double * d_matrix = nullptr;
    double * d_work = nullptr;
    int *d_info = nullptr;

    int info = 0;
    int lwork = 0;
    // double * eigenvalues_buffer = new double [n];
    double * eigenvectors_buffer = (double *)malloc(n*n*sizeof(double));
    // double * eigenvectors_buffer = new double [n*n];
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolver_handle));
    // CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    // CUSOLVER_CHECK(hipsolverSetStream(cusolver_handle, stream));

    CHECK(hipMalloc((void**)&d_matrix, n * n * sizeof(double)));
    CHECK(hipMalloc((void**)&d_eigenvalues,n*sizeof(double)));
    CHECK(hipMalloc((void**)&d_info,sizeof(int)));

    CHECK(hipMemcpy(d_matrix, matrix, n * n * sizeof(double), hipMemcpyHostToDevice));

    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    CUSOLVER_CHECK(hipsolverDnDsyevd_bufferSize(cusolver_handle,jobz,uplo,n,d_matrix,lda,d_eigenvalues,&lwork));

    CHECK(hipMalloc((void**)&d_work,sizeof(double)*lwork));

    // compute spectrum

    CUSOLVER_CHECK(hipsolverDnDsyevd(cusolver_handle, jobz, uplo, n, d_matrix, lda, d_eigenvalues, d_work, lwork, d_info));
    // check_cusolver(status);
    // std::printf("after syevd: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    int start = eigvals[0];
    int end = eigvals[1];
    int length = end - start + 1;
    CHECK(hipMemcpy(eigenvectors_buffer, d_matrix, n * n * sizeof(double), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(eigenvalues, d_eigenvalues+start, length*sizeof(double),hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost));
    
    CHECK(hipFree(d_matrix));
    CHECK(hipFree(d_eigenvalues));
    CHECK(hipFree(d_work));
    CHECK(hipFree(d_info));
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolver_handle));
    hipDeviceReset();
    // 
    for(int i = 0; i< length; i++){
        for(int j = 0; j< n; j++){
            eigenvectors[j*length+i] = eigenvectors_buffer[i*n+j];
        }
    }

    // delete [] eigenvectors_buffer;
    free(eigenvectors_buffer);
    

}





